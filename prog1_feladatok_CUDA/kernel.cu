#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <fstream>
#include <algorithm>

__global__ void mappingSquareKernel(unsigned short int* dest, const float* cReal, const float* cImag)
{
	int i = threadIdx.x + (blockIdx.x*blockDim.x);

	float zRealLoc = cReal[i];
	float zImagLoc = cImag[i];

	int k = 0;

	for (k = 0; k < 64; k++)
	{
		float temp = zRealLoc * zRealLoc - zImagLoc * zImagLoc + cReal[i];
		zImagLoc = 2.0 * zRealLoc * zImagLoc + cImag[i];
		zRealLoc = temp;

		if (fabs(zRealLoc) > 2.0 || fabs(zImagLoc) > 2.0)
			break;
	}

	dest[i] = k;
}

__global__ void mappingBiomKernel(unsigned short int* dest, const float* cReal, const float* cImag)
{
	int i = threadIdx.x + (blockIdx.x*blockDim.x);

	float zRealLoc = cReal[i];
	float zImagLoc = cImag[i];

	float dR = -0.8;
	float dI = 0.156;

	float vR = 0;
	float vI = 0;


	int k = 0;

	for (k = 0; k < 64; k++)
	{
		vR = zRealLoc * zRealLoc - zImagLoc * zImagLoc + dR;
		vI = 2.0 * zRealLoc * zImagLoc + dI;
		k++;

		zRealLoc = vR * vR - vI * vI + dR;
		zImagLoc = 2.0 * vR * vI + dI;

		if (fabs(zRealLoc) > 2.0 || fabs(zImagLoc) > 2.0)
			break;
	}

	dest[i] = k;
}

void mandelWithCuda(unsigned short int* iter, const float* bR, const float* bI, unsigned int WIDTH, unsigned int HEIGHT)
{
	unsigned int size = WIDTH * HEIGHT;

	float* dev_bR = 0;
	float* dev_bI = 0;
	unsigned short int* dev_iter = 0;

	hipError_t asd;

	hipSetDevice(0);

	asd = hipMalloc((void**)&dev_iter, size * sizeof(unsigned short));
	asd = hipMalloc((void**)&dev_bR, size * sizeof(float));
	asd = hipMalloc((void**)&dev_bI, size * sizeof(float));


	asd = hipMemcpy(dev_bR, bR, size * sizeof(float), hipMemcpyHostToDevice);
	asd = hipMemcpy(dev_bI, bI, size * sizeof(float), hipMemcpyHostToDevice);


	dim3 dimBlock = dim3(std::min(1024u, size));;
	dim3 dimGrid = dim3(std::ceil((float)size / 1024));

	mappingSquareKernel << <dimGrid, dimBlock >> > (dev_iter, dev_bR, dev_bI);

	asd = hipGetLastError();


	hipDeviceSynchronize();

	hipMemcpy(iter, dev_iter, size * sizeof(unsigned short), hipMemcpyDeviceToHost);
}

int main()
{
	const unsigned int WIDTH = 512;
	const unsigned int HEIGHT = 512;

	float* bReal = new float[WIDTH*HEIGHT];
	float* bImag = new float[WIDTH*HEIGHT];
	unsigned short int* iter = new unsigned short int[WIDTH*HEIGHT];

	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++)
		{
			bReal[i*HEIGHT + j] = (float)i / WIDTH * 4.0 - 2.0;
			bImag[i*HEIGHT + j] = (float)j / HEIGHT * 4.0 - 2.0;
		}


	mandelWithCuda(iter, bReal, bImag, WIDTH, HEIGHT);

	hipDeviceReset();

	delete[] bReal;
	delete[] bImag;

	std::ofstream out;
	out.open("out.pgm");

	out << "P2\n";
	out << HEIGHT << " " << WIDTH << "\n";
	out << "63\n";

	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++, out << "\n")
			out << iter[i*HEIGHT + j] << " ";

	out.close();

	//convert to png with 'magick out.pgm out.png'

	delete[] iter;
}