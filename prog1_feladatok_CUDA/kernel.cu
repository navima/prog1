#include "hip/hip_runtime.h"


#include <iostream>
#include <iomanip>
#include <fstream>
#include <algorithm>
#include <string>

__global__ void mappingSquareKernel(unsigned short int* dest, const float* cReal, const float* cImag)
{
	int i = threadIdx.x+(blockIdx.x*blockDim.x);

	float zRealLoc = cReal[i];
	float zImagLoc = cImag[i];

	int k = 0;

	for (k = 0; k < 64; k++)
	{
		float temp = zRealLoc * zRealLoc - zImagLoc * zImagLoc + cReal[i];
		zImagLoc = 2.0 * zRealLoc * zImagLoc + cImag[i];
		zRealLoc = temp;
		//k++;

		if (fabs(zRealLoc) > 2.0 || fabs(zImagLoc) > 2.0)
		{
			break;
		}
	}

	dest[i] = k;
	//dest[i] = i;
}

__global__ void mappingBiomKernel(unsigned short int* dest, const float* cReal, const float* cImag)
{
	int i = threadIdx.x + (blockIdx.x*blockDim.x);

	float zRealLoc = cReal[i];
	float zImagLoc = cImag[i];

	float dR = -0.8;
	float dI= 0.156;

	float vR = 0;
	float vI = 0;


	int k = 0;

	for (k = 0; k < 64; k++)
	{
		vR = zRealLoc * zRealLoc - zImagLoc * zImagLoc + dR;
		vI = 2.0 * zRealLoc * zImagLoc + dI;
		k++;

		zRealLoc = vR * vR - vI * vI + dR;
		zImagLoc = 2.0 * vR * vI + dI;

		if (fabs(zRealLoc) > 2.0 || fabs(zImagLoc) > 2.0)
		{
			break;
		}
	}

	dest[i] = k;
	//dest[i] = i;
}


void mandelWithCuda(unsigned short int* iter, const float* bR,const float* bI, unsigned int WIDTH, unsigned int HEIGHT)
{
	unsigned int size = WIDTH * HEIGHT;

	float* dev_bR = 0;
	float* dev_bI = 0;
	unsigned short int* dev_iter = 0;
	

	hipError_t asd;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output)
	asd =hipMalloc((void**)&dev_iter, size * sizeof(unsigned short));
	asd =hipMalloc((void**)&dev_bR, size * sizeof(float));
											  
	asd=hipMalloc((void**)&dev_bI, size * sizeof(float));


	// Copy input vectors from host memory to GPU buffers.
	asd=hipMemcpy(dev_bR, bR, size * sizeof(float), hipMemcpyHostToDevice);
										 
	asd=hipMemcpy(dev_bI, bI, size * sizeof(float), hipMemcpyHostToDevice);


	// Launch a kernel on the GPU with one thread for each element.

	dim3 dimBlock = dim3(std::min(1024u, size));;
	dim3 dimGrid = dim3(std::ceil((float)size / 1024));

	mappingSquareKernel <<<dimGrid, dimBlock>>> (dev_iter, dev_bR, dev_bI);

	// Check for any errors launching the kernel
	
	asd = hipGetLastError();

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(iter, dev_iter, size * sizeof(unsigned short), hipMemcpyDeviceToHost);
}

int main()
{
    const unsigned int WIDTH = 4096;
	const unsigned int HEIGHT = 4096;

    float* bReal = new float[WIDTH*HEIGHT];
	float* bImag = new float[WIDTH*HEIGHT];
	unsigned short int* iter = new unsigned short int[WIDTH*HEIGHT];

	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			bReal[i*HEIGHT + j] = (float)i / WIDTH * 4.0 - 2.0;
			bImag[i*HEIGHT + j] = (float)j / HEIGHT * 4.0 - 2.0;
			//std::cout << std::setw(6) << std::left << std::fixed <<std::setprecision(2)<<bImag[i*HEIGHT + j];
		}
		//std::cout << "\n";
	}


	mandelWithCuda(iter, bReal, bImag, WIDTH, HEIGHT);

	hipDeviceReset();


	std::ofstream out;
	out.open("out.pgm");

	out << "P2\n";
	out << HEIGHT << " " << WIDTH << "\n";
	out << "63\n";

	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			//std::cout << std::setw(4) << std::left << iter[i*HEIGHT + j];
			out << iter[i*HEIGHT + j] << " ";//(unsigned char)iter[i*WIDTH + j];
			//out <<std::setw(4)<<std::left<< iter[i*HEIGHT+j] <<" ";
		}
		//std::cout << "\n";
		out << "\n";
	}

	out.close();

	delete[] bReal;
	delete[] bImag;
	delete[] iter;
}